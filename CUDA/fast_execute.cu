#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <iostream>

#define ITERATIONS 104000
#define BATCH_SIZE 2097152
#define MAX_REGISTER 5
#define SEED 314159

#define MAX_ZEROES 3
#define MAX_TRANSFERS 7
#define MAX_JUMPS 4

#define MAX_INSTRUCTIONS 150
#define PROGRAM_LINES 10

#define PROGRAM_SIZE (40 * sizeof(unsigned char))
#define BLOCK_SIZE 256

#define TOTAL_BLOCKS (BATCH_SIZE / BLOCK_SIZE)
#define TOTAL_PROGRAMS_MEMORY (PROGRAM_SIZE * BATCH_SIZE * sizeof(unsigned char))
#define TOTAL_RESULTS_MEMORY (BATCH_SIZE * sizeof(unsigned char))
#define TOTAL_RANDOM_STATE_SIZE (BATCH_SIZE * sizeof(hiprandState))

__global__
void initialize_states(hiprandState *states) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  hiprand_init(SEED, index, 0, &states[index]);
}

#define random(min, max) ((unsigned char)truncf(hiprand_uniform(&state) * (max - min + 0.999999f) + min))

__global__
void compute_program(int n, int it, unsigned char *programs, unsigned char *results, hiprandState *states, unsigned short *executedInstructions) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  // Get local copy
  unsigned char P[PROGRAM_SIZE];
  hiprandState state = states[index];
  memcpy(P, &programs[PROGRAM_SIZE * index], PROGRAM_SIZE);

  // Generate program
  int zeroes = MAX_ZEROES;
  int jumps = MAX_JUMPS;
  int transfers = MAX_TRANSFERS;

  for (int i = 0; i < PROGRAM_LINES; i++) {
    if (i + 1 == PROGRAM_LINES)
      zeroes = 0;

    if (zeroes == 0) {
      if (jumps == 0) {
        if (transfers == 0) {
          P[i * 4] = 0;
        } else {
          P[i * 4] = random(1, 2);
        }
      } else {
        if (transfers == 0) {
          P[i * 4] = 1 + random(0, 1) * 2;
        } else {
          P[i * 4] = random(1, 3);
        }
      }
    } else {
      if (jumps == 0) {
        if (transfers == 0) {
          P[i * 4] = random(0, 1);
        } else {
          P[i * 4] = random(0, 2);
        }
      } else {
        if (transfers == 0) {
          P[i * 4] = random(0, 2);

          // Map 2 -> 3
          if (P[i * 4] == 2)
            P[i * 4]++;
        } else {
          P[i * 4] = random(0, 3);
        }
      }
    }

    if (P[i * 4] == 0) {
      zeroes--;
      P[i * 4 + 1] = random(1, MAX_REGISTER);
    } else if (P[i * 4] == 1) {
      P[i * 4 + 1] = random(1, MAX_REGISTER);
    } else if (P[i * 4] == 2) {
      transfers--;
      P[i * 4 + 1] = random(1, MAX_REGISTER);
      P[i * 4 + 2] = random(2, MAX_REGISTER);

      if (P[i * 4 + 1] == P[i * 4 + 2]) {
        P[i * 4 + 2] = 1;
      }      
    } else if (P[i * 4] == 3) {
      jumps--;

      P[i * 4 + 1] = random(1, MAX_REGISTER);
      P[i * 4 + 2] = random(1, MAX_REGISTER);
      P[i * 4 + 3] = random(1, PROGRAM_LINES + 1);
    }
  }

  // Execution
  unsigned short count = 0;
  int ip = 0;
  int R[MAX_REGISTER];

  for (int i = 0; i < MAX_REGISTER; i++)
    R[i] = 0;

  while ((0 <= ip && ip < PROGRAM_LINES) && count < MAX_INSTRUCTIONS) {
    count++;

    int kind = P[ip * 4 + 0];
    int p1 = P[ip * 4 + 1];
    int p2 = P[ip * 4 + 2];
    int p3 = P[ip * 4 + 3];

    if (kind == 0)
      R[p1 - 1] = 0;
    else if (kind == 1)
      R[p1 - 1]++;
    else if (kind == 2)
      R[p2 - 1] = R[p1 - 1];
    else if (kind == 3 && (R[p1 - 1] == R[p2 - 1])) {
      ip = p3 - 1;
      continue;
    }
      
    ip += 1;
  }

  if (count < MAX_INSTRUCTIONS) {
    results[index] = R[0];
    executedInstructions[index] = count;
  }

  // Reload memory
  states[index] = state;
  memcpy(&programs[PROGRAM_SIZE * index], P, PROGRAM_SIZE);
}

void print_program(unsigned char *program) {
  for (int i = 0; i < PROGRAM_LINES; i++) {
    int kind = program[i * 4 + 0];
    int p1 = program[i * 4 + 1];
    int p2 = program[i * 4 + 2];
    int p3 = program[i * 4 + 3];

    if (kind == 0)
      printf("Z(%d)\n", p1);
    else if (kind == 1)
      printf("S(%d)\n", p1);
    else if (kind == 2)
      printf("T(%d,%d)\n", p1, p2);
    else if (kind == 3)
      printf("J(%d,%d,%d)\n", p1, p2, p3);
  }
}

int main() {
  unsigned char bestProgram[PROGRAM_SIZE];
  unsigned char bestProgramResult = 0;
  unsigned short bestExecutedInstructions = 1000;

  hiprandState *randomStates;
  unsigned char *programs, *results;
  unsigned short *executedInstructions;

  hipMalloc(&randomStates, TOTAL_RANDOM_STATE_SIZE);
  hipMallocManaged(&programs, TOTAL_PROGRAMS_MEMORY);
  hipMallocManaged(&results, TOTAL_RESULTS_MEMORY);
  hipMallocManaged(&executedInstructions, BATCH_SIZE * sizeof(unsigned short));

  int device;
  hipGetDevice(&device);

  initialize_states<<<TOTAL_BLOCKS, BLOCK_SIZE>>>(randomStates);

  for (int i = 0; i < ITERATIONS; i++) {
    hipMemPrefetchAsync(programs, TOTAL_PROGRAMS_MEMORY, device, NULL);
    hipMemPrefetchAsync(results, TOTAL_RESULTS_MEMORY, device, NULL);
    hipMemPrefetchAsync(executedInstructions, BATCH_SIZE * sizeof(unsigned short), device, NULL);

    compute_program<<<TOTAL_BLOCKS, BLOCK_SIZE>>>(BATCH_SIZE, i, programs, results, randomStates, executedInstructions);
    hipDeviceSynchronize();

    for (int j = 0; j < BATCH_SIZE; j++) {
      if (results[j] > bestProgramResult || (results[j] == bestProgramResult && executedInstructions[j] < bestExecutedInstructions)) {
        hipMemcpy(bestProgram, &programs[j * PROGRAM_SIZE], PROGRAM_SIZE, hipMemcpyDeviceToHost);
        bestProgramResult = results[j];
        bestExecutedInstructions = executedInstructions[j];

        printf("Better program found: %2d in %3d instructions (@%d[%d])\n", bestProgramResult, bestExecutedInstructions, i, j);
        print_program(bestProgram);
      }

    }
  }

  hipFree(randomStates);
  hipFree(programs);
  hipFree(results);
}